#include "hip/hip_runtime.h"
#include "common.h"
#include "physics.h"
#include <vector>
#include "image.h"
#include "CycleTimer.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

__global__ void kernelSimSteps(){
    return;
}
 __global__ void kernelRenderImage(){
    return;
}

void simulateStepCuda(std::vector<Node>& new_nodes,
                      std::vector<Node>& nodes,
                      const stepParams params){
    /* uh i don't think nodes are going to fit into the warps here. Like I'm
     * pretty sure we'll have to move the properties to some array and do
     * with that instead of just passing the structs into the device memory.
     * It might just be easier to do just the visualizer on this. 
     */
    return;
}

void visualizer(std::vector<Node> & nodes){
    return;
}